#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>

#include <stdio.h>

static inline void check(hipError_t err, const char *context) {
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA error at %s: %s\n", context, hipGetErrorString(err));
    std::exit(EXIT_FAILURE);
  }
}

#define CHECK(x) check(x, #x)

__global__ void saxpy(int n, float a, float *x, float *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    y[i] = a * x[i] + y[i];
}

int main(void) {
  setbuf(stderr, NULL);

  int rtVersion, driverVersion;
  CHECK(hipRuntimeGetVersion(&rtVersion));
  CHECK(hipDriverGetVersion(&driverVersion));

  fprintf(stderr, "Runtime version: %d\n", rtVersion);
  fprintf(stderr, "Driver version: %d\n", driverVersion);

  constexpr int N = 1 << 20;

  std::vector<float> xHost(N), yHost(N);
  for (int i = 0; i < N; i++) {
    xHost[i] = 1.0f;
    yHost[i] = 2.0f;
  }

  fflush(stderr);

  float *xDevice, *yDevice;
  CHECK(hipMalloc(&xDevice, N * sizeof(float)));
  CHECK(hipMalloc(&yDevice, N * sizeof(float)));

  CHECK(hipMemcpy(xDevice, xHost.data(), N * sizeof(float),
                   hipMemcpyHostToDevice));
  CHECK(hipMemcpy(yDevice, yHost.data(), N * sizeof(float),
                   hipMemcpyHostToDevice));

  saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, xDevice, yDevice);
  CHECK(hipGetLastError());

  CHECK(hipMemcpy(yHost.data(), yDevice, N * sizeof(float),
                   hipMemcpyDeviceToHost));

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(yHost[i] - 4.0f));
  fprintf(stderr, "Max error: %f\n", maxError);

  CHECK(hipFree(xDevice));
  CHECK(hipFree(yDevice));
}
